#include "hip/hip_runtime.h"
/*
 * HelperFunctions.cpp
 *
 *  Created on: 22/08/2014
 *      Author: vincentvillani
 */

#include "HelperFunctions.h"

//TODO: ADD CEIL CALLS TO BLOCKNUM CALCULATIONS


//vectorLength == binSize of the stokes vectors
//powerOfTwoVectorLength has to be a power of two
//evenResultGridDim has to be an even number
//d_lhsVectorLength == d_rhsVectorLength == powerOfTwoVectorLength
//TODO: TO MANY RESTRICTIONS? WILL PROBABLY STILL WORK IF powerOfTwoVectorLength / evenResultGridDim == an even number
void computeUpperTriangularOuterProduct(float* d_resultMatrix, int resultMatrixLength, float* d_lhsVector,
		float* d_rhsVector, int powerOfTwoVectorLength, int evenResultGridDim, int threadNum)
{
	if(evenResultGridDim % 2 != 0)
	{
		printf("Error: computeUpperTriangularOuterProduct() param 'evenResultGridDim' expects an even number");
		return;
	}

	//calculate number of cuda blocks needed for each kernel
	int cudaWholeOuterProductBlockNum = max((float)1,  min( ceil((powerOfTwoVectorLength * powerOfTwoVectorLength) / threadNum), (float)(1 << 16) - 1));
	int cudaUpperTriOuterProductBlockNum = max((float)1,  min( ceil(((powerOfTwoVectorLength * (powerOfTwoVectorLength + 1) / 2)) / threadNum), (float)(1 << 16) - 1));


	//for every 'block' in the result matrix
	for(int i = 0; i < evenResultGridDim; ++i)
	{
		//call upper triangular outer product on along the diagonal
		upperTrianglarOuterProductSum<<<cudaUpperTriOuterProductBlockNum, threadNum>>>
				(d_resultMatrix, d_lhsVector, d_rhsVector,powerOfTwoVectorLength, evenResultGridDim, i);

		//call the whole outer product kernel for the remaining blocks on this row
		for(int j = i + 1; j < evenResultGridDim; ++j)
		{
			wholeOuterProductSum<<<cudaWholeOuterProductBlockNum, threadNum>>>
					(d_resultMatrix, d_lhsVector, d_rhsVector, powerOfTwoVectorLength, evenResultGridDim, i, j);
		}
	}

	/*
	//check for errors
	hipError_t error2 = hipDeviceSynchronize();

	if(error2 != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error2));
		return;
	}
	*/

	//DEBUG - print results
	//copyAndPrint(d_resultMatrix, resultMatrixLength, evenResultGridDim * powerOfTwoVectorLength);

}


/*
//vectorLength == binSize of the stokes vectors
//powerOfTwoVectorLength has to be a power of two
//evenResultGridDim has to be an even number
//d_lhsVectorLength == d_rhsVectorLength == powerOfTwoVectorLength
//TODO: TO MANY RESTRICTIONS? WILL PROBABLY STILL WORK IF powerOfTwoVectorLength / evenResultGridDim == an even number
void computeUpperTriangularOuterProductStream(float* d_resultMatrix, int resultMatrixLength, float* d_lhsVector,
		float* d_rhsVector, int powerOfTwoVectorLength, int evenResultGridDim, int threadNum, hipStream_t* stream1,
		hipStream_t* stream2)
{
	if(evenResultGridDim % 2 != 0)
	{
		printf("Error: computeUpperTriangularOuterProduct() param 'evenResultGridDim' expects an even number");
		return;
	}

	//calculate the number of kernels for each stream
	bool evenStream = false;

	//calculate number of cuda blocks needed for each kernel
	int cudaWholeOuterProductBlockNum = max(1,  min( (powerOfTwoVectorLength * powerOfTwoVectorLength) / threadNum, (1 << 16) - 1));
	int cudaUpperTriOuterProductBlockNum = max(1,  min( ((powerOfTwoVectorLength * (powerOfTwoVectorLength + 1) / 2)) / threadNum, (1 << 16) - 1));


	//for every 'block' in the result matrix
	for(int i = 0; i < evenResultGridDim; ++i)
	{
		if(evenStream)
		{
			//call upper triangular outer product on along the diagonal
			upperTrianglarOuterProductSum<<<cudaUpperTriOuterProductBlockNum, threadNum, 0, *stream1>>>
					(d_resultMatrix, d_lhsVector, d_rhsVector,powerOfTwoVectorLength, evenResultGridDim, i);
		}
		else
		{
			//call upper triangular outer product on along the diagonal
			upperTrianglarOuterProductSum<<<cudaUpperTriOuterProductBlockNum, threadNum, 0, *stream2>>>
					(d_resultMatrix, d_lhsVector, d_rhsVector,powerOfTwoVectorLength, evenResultGridDim, i);
		}

		evenStream = !evenStream; //switch stream

		//call the whole outer product kernel for the remaining blocks on this row
		for(int j = i + 1; j < evenResultGridDim; ++j)
		{
			if(evenStream)
			{
				wholeOuterProductSum<<<cudaWholeOuterProductBlockNum, threadNum, 0, *stream1>>>
						(d_resultMatrix, d_lhsVector, d_rhsVector, powerOfTwoVectorLength, evenResultGridDim, i, j);
			}
			else
			{
				wholeOuterProductSum<<<cudaWholeOuterProductBlockNum, threadNum, 0, *stream2>>>
						(d_resultMatrix, d_lhsVector, d_rhsVector, powerOfTwoVectorLength, evenResultGridDim, i, j);
			}

			evenStream = !evenStream; //switch stream

		}
	}


	//check for errors
	hipError_t error2 = hipDeviceSynchronize();

	if(error2 != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error2));
		return;
	}


	//DEBUG - print results
	//copyAndPrint(d_resultMatrix, resultMatrixLength, evenResultGridDim * powerOfTwoVectorLength);

}
*/



//vectorLength == binSize of the stokes vectors
//powerOfTwoVectorLength has to be a power of two
//evenResultGridDim has to be an even number
//d_lhsVectorLength == d_rhsVectorLength == powerOfTwoVectorLength
//TODO: TO MANY RESTRICTIONS? WILL PROBABLY STILL WORK IF powerOfTwoVectorLength / evenResultGridDim == an even number
void computeUpperTriangularOuterProductOneBigKernel(float*  d_resultMatrix, int resultMatrixLength, float*  d_lhsVector,
		int powerOfTwoVectorLength, int threadNum)
{
	//calculate number of cuda blocks needed for each kernel
	int cudaUpperTriOuterProductBlockNum = max((float)1,  min( ceil(upperTriangularLength(powerOfTwoVectorLength) / threadNum), (float)(1 << 16) - 1));


	//call upper triangular outer product on along the diagonal
	upperTrianglarOuterProductSumOneBigKernel<<<cudaUpperTriOuterProductBlockNum, threadNum>>>
			(d_resultMatrix, d_lhsVector, powerOfTwoVectorLength);


	/*
	//check for errors
	hipError_t error2 = hipDeviceSynchronize();

	if(error2 != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error2));
		return;
	}

*/
	//DEBUG - print results
	//copyAndPrint(d_resultMatrix, resultMatrixLength, powerOfTwoVectorLength);


}



void arbTest(int vectorLength, int resultGridDim)
{
	int resultMatrixLength = upperTriangularLength(vectorLength * resultGridDim);

	float* h_vector;

	float* d_resultMatrix;
	float* d_vector;

	h_vector = (float*)malloc(sizeof(float) * vectorLength);


	hipMalloc(&d_resultMatrix, sizeof(float) * resultMatrixLength);
	hipMalloc(&d_vector, sizeof(float) * vectorLength);

	hipMemset(d_resultMatrix, 0, sizeof(float) * resultMatrixLength);


	for(int i = 0; i < vectorLength; ++i)
	{
		h_vector[i] = i + 1;
	}

	hipMemcpy(d_vector, h_vector, sizeof(float) * vectorLength, hipMemcpyHostToDevice);

	computeUpperTriangularOuterProduct(d_resultMatrix, resultMatrixLength, d_vector, d_vector, vectorLength, resultGridDim, 256);

	free(h_vector);

	hipFree(d_resultMatrix);
	hipFree(d_vector);

}



void arbTestOneBigKernel(int vectorLength)
{
	int resultMatrixLength = upperTriangularLength(vectorLength);

	float* h_vector;

	float* d_resultMatrix;
	float* d_vector;

	h_vector = (float*)malloc(sizeof(float) * vectorLength);


	hipMalloc(&d_resultMatrix, sizeof(float) * resultMatrixLength);
	hipMalloc(&d_vector, sizeof(float) * vectorLength);

	hipMemset(d_resultMatrix, 0, sizeof(float) * resultMatrixLength);


	for(int i = 0; i < vectorLength; ++i)
	{
		h_vector[i] = i + 1;
	}

	hipMemcpy(d_vector, h_vector, sizeof(float) * vectorLength, hipMemcpyHostToDevice);

	computeUpperTriangularOuterProductOneBigKernel(d_resultMatrix, resultMatrixLength, d_vector, vectorLength, 256);

	free(h_vector);

	hipFree(d_resultMatrix);
	hipFree(d_vector);

}


void printResultUpperTriangular(float* result, int rowLength, bool genFile)
{
	int numZeros = 0;
	int iterator = 0;

	if(genFile)
	{
		FILE* file = fopen("/mnt/home/vvillani/deviceOuterProductFinal/resultMatrix.txt", "w");

		//for every row
		for(int i = 0; i < rowLength; ++i)
		{
			//print preceding zeros
			for(int j = 0; j < numZeros; ++j)
			{
				fprintf(file, "0, ");
			}

			//print array values
			for(int k = 0; k < rowLength - numZeros; ++k)
			{
				fprintf(file, "%d, ", (int)result[iterator]);
				++iterator;
			}

			fprintf(file, "\n");
			numZeros++;
		}

	}

	numZeros = 0;
	iterator = 0;

	//for every row
	for(int i = 0; i < rowLength; ++i)
	{
		//print preceding zeros
		for(int j = 0; j < numZeros; ++j)
		{
			printf("0, ");
		}

		//print array values
		for(int k = 0; k < rowLength - numZeros; ++k)
		{
			printf("%d, ", (int)result[iterator]);
			++iterator;
		}

		printf("\n");
		numZeros++;
	}

	printf("\n------------------------\n");

}



void copyAndPrint(float* deviceData, int arrayLength, int rowLength)
{
	float* hostData = (float*)malloc(sizeof(float) * arrayLength);
	hipMemcpy(hostData, deviceData, sizeof(float) * arrayLength, hipMemcpyDeviceToHost);
	printResultUpperTriangular(hostData, rowLength, false);
}


void setCPUTimer(clock_t* timer)
{
    *timer = clock();
}


double calcCPUTime(clock_t startTime, clock_t endTime)
{
    return (double)(endTime - startTime) / CLOCKS_PER_SEC;
}



void runBenchmark(int iterations)
{
	float* h_vector;

	float* d_resultMatrix;
	float* d_vector;


	clock_t timers[2]; //start and end timers for all 6 bin sizes
	double timingResult; //total elapsed time for each bin size benchmark

	FILE* file = fopen("/mnt/home/vvillani/deviceOuterProductFinal/BenchmarkResults.txt", "w");

	int resultGridDim = 4;
	int binSize;
	int threadSize;
	//const int iterations = 3000;
	int resultMatrixLength;

	fprintf(file, "ITERATIONS: %d\n\n", iterations);

	//for each bin size - 128 to 4096
	for(int i = 0; i < 6; ++i)
	{
		binSize = 1 << (7 + i);

		fprintf(file, "\n\n\nBINSIZE: %d\n\n", binSize);

		h_vector = (float*)malloc(sizeof(float) * binSize);

		resultMatrixLength = upperTriangularLength(binSize * resultGridDim);
		hipMalloc(&d_resultMatrix, sizeof(float) * resultMatrixLength);
		hipMalloc(&d_vector, sizeof(float) * binSize);

		hipMemset(d_resultMatrix, 0, sizeof(float) * resultMatrixLength);

		for(int k = 0; k < binSize; ++k)
			h_vector[k] = k + 1;

		hipMemcpy(d_vector, h_vector, sizeof(float) * binSize, hipMemcpyHostToDevice);

		//for each threadSize - 64 to 1024
		for(int j = 0; j < 5; ++j)
		{
			threadSize = 1 << (6 + j);

			setCPUTimer(&timers[0]); //start time

			//perform the benchmark iteration times
			for(int z = 0; z < iterations; ++z)
			{
				computeUpperTriangularOuterProduct(d_resultMatrix, resultMatrixLength, d_vector, d_vector, binSize, resultGridDim, threadSize);
			}

			hipDeviceSynchronize(); //wait till all kernels are finished
			setCPUTimer(&timers[1]); //end time
			timingResult = calcCPUTime(timers[0], timers[1]); //result

			//write the result to the file
			fprintf(file, "THREADSIZE %d: %f\n", threadSize, timingResult);

		}



		free(h_vector);
		hipFree(d_resultMatrix);
		hipFree(d_vector);

		printf("Finished iteration %d\n", i);
	}

	fclose(file);
}



void runBenchmarkOneBigKernel(int iterations)
{
	float* h_vector;

	float*  d_resultMatrix;
	float*  d_vector;


	clock_t timers[2]; //start and end timers for all 6 bin sizes
	double timingResult; //total elapsed time for each bin size benchmark

	FILE* file = fopen("/mnt/home/vvillani/deviceOuterProductFinal/BenchmarkResults.txt", "w");

	int resultGridDim = 1;
	int binSize;
	int threadSize;
	//const int iterations = 3000;
	int resultMatrixLength;

	fprintf(file, "ITERATIONS: %d\n\n", iterations);

	//for each bin size - 128 to 4096
	for(int i = 0; i < 6; ++i)
	{
		binSize = (1 << (7 + i)) * 4; //4 stokes vectors

		fprintf(file, "\n\n\nBINSIZE: %d\n\n", binSize / 4);

		h_vector = (float*)malloc(sizeof(float) * binSize);

		resultMatrixLength = upperTriangularLength(binSize * resultGridDim);
		hipMalloc(&d_resultMatrix, sizeof(float) * resultMatrixLength);
		hipMalloc(&d_vector, sizeof(float) * binSize);

		hipMemset(d_resultMatrix, 0, sizeof(float) * resultMatrixLength);

		for(int k = 0; k < binSize; ++k)
			h_vector[k] = k + 1;

		hipMemcpy(d_vector, h_vector, sizeof(float) * binSize, hipMemcpyHostToDevice);

		//for each threadSize - 64 to 1024
		for(int j = 0; j < 5; ++j)
		{
			threadSize = 1 << (6 + j);

			setCPUTimer(&timers[0]); //start time

			//perform the benchmark iteration times
			for(int z = 0; z < iterations; ++z)
			{
				computeUpperTriangularOuterProductOneBigKernel(d_resultMatrix, resultMatrixLength, d_vector, binSize, threadSize);
			}

			hipDeviceSynchronize(); //wait till all kernels are finished
			setCPUTimer(&timers[1]); //end time
			timingResult = calcCPUTime(timers[0], timers[1]); //result

			//write the result to the file
			fprintf(file, "THREADSIZE %d: %f\n", threadSize, timingResult);

		}



		free(h_vector);
		hipFree(d_resultMatrix);
		hipFree(d_vector);

		printf("Finished iteration %d\n", i);
	}

	fclose(file);
}

/*
void runBenchmarkStreams(int iterations)
{
	float* h_vector;

	float* d_resultMatrix;
	float* d_vector;

	hipStream_t stream1;
	hipStream_t stream2;

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	clock_t timers[2]; //start and end timers for all 6 bin sizes
	double timingResult; //total elapsed time for each bin size benchmark

	FILE* file = fopen("/mnt/home/vvillani/deviceOuterProductFinal/BenchmarkResults.txt", "w");

	int resultGridDim = 4;
	int binSize;
	int threadSize;
	//const int iterations = 3000;
	int resultMatrixLength;

	fprintf(file, "ITERATIONS: %d\n\n", iterations);

	//for each bin size - 128 to 4096
	for(int i = 0; i < 6; ++i)
	{
		binSize = 1 << (7 + i);

		fprintf(file, "\n\n\nBINSIZE: %d\n\n", binSize);

		h_vector = (float*)malloc(sizeof(float) * binSize);

		resultMatrixLength = upperTriangularLength(binSize * resultGridDim);
		hipMalloc(&d_resultMatrix, sizeof(float) * resultMatrixLength);
		hipMalloc(&d_vector, sizeof(float) * binSize);

		hipMemset(d_resultMatrix, 0, sizeof(float) * resultMatrixLength);

		for(int k = 0; k < binSize; ++k)
			h_vector[k] = k + 1;

		hipMemcpy(d_vector, h_vector, sizeof(float) * binSize, hipMemcpyHostToDevice);

		//for each threadSize - 64 to 1024
		for(int j = 0; j < 5; ++j)
		{
			threadSize = 1 << (6 + j);

			setCPUTimer(&timers[0]); //start time

			//perform the benchmark iteration times
			for(int z = 0; z < iterations; ++z)
			{
				computeUpperTriangularOuterProductStream(d_resultMatrix, resultMatrixLength, d_vector, d_vector, binSize, resultGridDim, threadSize, &stream1, &stream2);
			}

			hipDeviceSynchronize(); //wait till all kernels are finished
			setCPUTimer(&timers[1]); //end time
			timingResult = calcCPUTime(timers[0], timers[1]); //result

			//write the result to the file
			fprintf(file, "THREADSIZE %d: %f\n", threadSize, timingResult);

		}



		free(h_vector);
		hipFree(d_resultMatrix);
		hipFree(d_vector);

		printf("Finished iteration %d\n", i);
	}

	fclose(file);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
}

*/

int  upperTrianglarRowIndexIntrinsicHost(int idx, int matDim)
{
	int temp = matDim * (matDim + 1) / 2 - 1 - idx;
	int k = floorf( (sqrtf(8 * temp + 1) - 1) / 2);
	return matDim - 1 - k;
}


void testSqrt()
{
	int* h_indexes;
	int* d_indexes;

	int* h_results;
	int* h_gpuResults;

	int* d_results;

	int nCol = 4096 * 4;
	int num = upperTriangularLength(nCol);


	h_indexes = (int*)malloc(sizeof(int) * num);
	h_results = (int*)malloc(sizeof(int) * num);
	h_gpuResults = (int*)malloc(sizeof(int) * num);

	hipMalloc(&d_indexes, sizeof(int) * num);
	hipMalloc(&d_results, sizeof(int) * num);

	for(int i = 0; i < num; ++i)
		h_indexes[i] = i;

	hipMemcpy(d_indexes, h_indexes, sizeof(int) * num, hipMemcpyHostToDevice);

	squareRootIntrinsic<<< ceilf( num / 256), 256>>>(d_indexes, d_results, nCol, num);

	for(int i = 0; i < num; ++i)
		h_results[i] = upperTrianglarRowIndexIntrinsicHost(i, nCol);

	hipMemcpy(h_gpuResults, d_results, sizeof(int) * num, hipMemcpyDeviceToHost);

	//compare results

	for(int i = 0; i < num; ++i)
	{
		if(h_results[i] != h_gpuResults[i])
		{
			printf("ERORR: CPU %d, GPU %d\n", h_results[i], h_gpuResults[i]);
		}
	}

	printf("Test complete!\n");

	free(h_indexes);
	free(h_results);
	free(h_gpuResults);

	hipFree(d_indexes);
	hipFree(d_results);

}



